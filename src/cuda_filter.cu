#include "hip/hip_runtime.h"
#include "cuda_filter.h"
#include <hip/hip_runtime.h> // Ensure you include the CUDA runtime header

__global__ void process_cuda_kernel(float *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] += 10;  // Example operation
    }
}

void process_data_cuda(float *data, int size) {
    float *d_data;
    hipMalloc(&d_data, size * sizeof(float));
    hipMemcpy(d_data, data, size * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch kernel (for example, with 256 threads per block)
    int blocks = (size + 255) / 256;
    process_cuda_kernel<<<blocks, 256>>>(d_data, size);
    
    hipMemcpy(data, d_data, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_data);
}
